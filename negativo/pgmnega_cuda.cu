#include "hip/hip_runtime.h"
#include <stdio.h>
#include "pgmlib.h"

__global__ void kernel(int * device_mem, int lin, int cols, int maxval)
{
	int index = blockIdx.x * cols + threadIdx.x;

	device_mem[index] = maxval - device_mem[index];
}

int negative_cuda(PGM_t *pgm)
{
	int *device_mem;
	int size_bytes = sizeof(int) * pgm->rows * pgm->cols;

	hipMalloc ((void **) &device_mem, size_bytes);

	hipMemcpy ((void *) device_mem, (void *)pgm->image, size_bytes, hipMemcpyHostToDevice);

    kernel<<<pgm->rows, pgm->cols>>>(device_mem, pgm->rows, pgm->cols, pgm->maxval);

    hipMemcpy ((void *)pgm->image, (void *) device_mem, size_bytes, hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    return 0;
}
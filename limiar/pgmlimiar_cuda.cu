#include "hip/hip_runtime.h"
#include <stdio.h>
#include "pgmlib.h"

__global__ void kernel(int * device_mem, float limiar, int lin, int cols, int maxval)
{
	int index = blockIdx.x * cols + threadIdx.x;

	if (device_mem[index] >= 0.5 * maxval)
		device_mem[index] = maxval;
	else
		device_mem[index] = 0;
}

int limiar_cuda(PGM_t *pgm, float limiar)
{
	int *device_mem;
	int size_bytes = sizeof(int) * pgm->rows * pgm->cols;

	hipMalloc ((void **) &device_mem, size_bytes);

	hipMemcpy ((void *) device_mem, (void *)pgm->image, size_bytes, hipMemcpyHostToDevice);

    kernel<<<pgm->rows, pgm->cols>>>(device_mem, limiar, pgm->rows, pgm->cols, pgm->maxval);

    hipMemcpy ((void *)pgm->image, (void *) device_mem, size_bytes, hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    return 0;
}
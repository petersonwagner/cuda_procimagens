#include "hip/hip_runtime.h"
#include <stdio.h>
#include "pgmlib.h"

__global__ void kernel(int * d_in, int * d_out, int lin, int cols, int maxval)
{
	int index = blockIdx.x * cols + threadIdx.x;
	int sum = 0;

	//interior da matriz
	if(blockIdx.x > 1 && threadIdx.x > 1 && blockIdx.x < lin-2 && threadIdx.x < cols-2)
	{
		sum += d_in[(blockIdx.x-1) * cols + threadIdx.x-1];
		sum += d_in[(blockIdx.x-1) * cols + threadIdx.x  ];
		sum += d_in[(blockIdx.x-1) * cols + threadIdx.x+1];
		sum += d_in[ blockIdx.x    * cols + threadIdx.x-1];
		sum += d_in[ blockIdx.x    * cols + threadIdx.x  ];
		sum += d_in[ blockIdx.x    * cols + threadIdx.x+1];
		sum += d_in[(blockIdx.x+1) * cols + threadIdx.x-1];
		sum += d_in[(blockIdx.x+1) * cols + threadIdx.x  ];
		sum += d_in[(blockIdx.x+1) * cols + threadIdx.x+1];

		d_out[index] = sum / 9;
	}
}

int media_cuda(PGM_t *pgm, PGM_t *pgm_result)
{
	int *d_in, *d_out;
	int size_bytes = sizeof(int) * pgm->rows * pgm->cols;

	hipMalloc ((void **) &d_in,  size_bytes);
	hipMalloc ((void **) &d_out, size_bytes);

	hipMemset(d_out, 0, size_bytes);

	hipMemcpy ((void *) d_in, (void *) pgm->image, size_bytes, hipMemcpyHostToDevice);

    kernel<<<pgm->rows, pgm->cols>>>(d_in, d_out, pgm->rows, pgm->cols, pgm->maxval);

    hipMemcpy ((void *)pgm_result->image, (void *) d_out, size_bytes, hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    return 0;
}